#include "benchmark.h"
#include "wrapper.h"
#include <exception>
#include <yao/cuda/cuda_error.h>
#include <yao/io/binary_io.h>
#include <yao/cusp_ext/convert.h>
#include <yao/time/timer.h>
#include <cusp/csr_matrix.h>

#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>

#include <algorithm>
#include <numeric>

#include <cusp/system/tbb/detail/par.h>

#ifdef ENABLE_VTUNE
#include "vtune.h"
#endif

bool is_file_exist(std::string filename)
{
	std::ifstream infile(filename);
	return infile.good();
}

void read_matrix(const matrix_info_t &matrix, const std::vector<std::string> path,
	cusp::csr_matrix<int, float, cusp::host_memory> &csr_matrix, float ValueThreshold)
{
	std::string filename;
	bool is_found = false;
	for (auto &dir : path)
	{
		filename = dir + matrix.data_filename;
		if (is_file_exist(filename))
		{
			is_found = true;
			break;
		}
	}

	std::cout << "Read " << filename << std::endl;

	if (!is_found)
		throw std::runtime_error(filename + " NOT found!");

	std::vector<float> raw_data;
	yao::io::ReadBinaryArray(filename, raw_data);
	yao::cusp_ext::convert(&raw_data[0], matrix.num_rows, matrix.num_cols, csr_matrix, ValueThreshold);

	float nnz_percent = static_cast<float>(csr_matrix.num_entries) / (csr_matrix.num_rows * csr_matrix.num_cols);
	std::cout << "nnz%=" << nnz_percent << std::endl;
	
}


template <typename callable_t>
double time_spmv(callable_t c, int num_iterations)
{
	auto t1 = std::chrono::high_resolution_clock::now();
	for (int i = 0; i < num_iterations; i++)
		c();
	auto t2 = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double, std::milli> time = t2 - t1;

	return static_cast<double>(time.count()) / num_iterations;
}

// TODO Reduce redundant code
benchmark_result_t start_benchmark(benchmark_setting_t &setting)
{
	benchmark_result_t b_result;

	for (auto & matrix : setting.matrices)
	{
		
		matrix_result_t m_result(matrix);




		// CPU SpMV
		cusp::csr_matrix<int, float, cusp::host_memory> csr_matrix;
		read_matrix(matrix, setting.matrix_path, csr_matrix, setting.value_threshold);
		std::vector<float> x(matrix.num_cols, 1);
		std::vector<float> y(matrix.num_rows, 0);
		
		m_result.time.emplace(static_cast<std::string>("mkl"), 
			time_spmv([&csr_matrix, &x, &y] {blas::mkl::spmv(csr_matrix, x, y); }, setting.num_iterations));


		::cusp::array1d_view<typename std::vector<float>::iterator> x_view(x.begin(), x.end());
		::cusp::array1d_view<typename std::vector<float>::iterator> y_view(y.begin(), y.end());
		//CPU cusp



		#ifdef ENABLE_VTUNE
		__itt_resume();
		__itt_task_begin(domain, __itt_null, __itt_null, op_task);
		#endif

		m_result.time.emplace(static_cast<std::string>("cusp_tbb"),
			time_spmv(
				[&csr_matrix, &x_view, &y_view]
		{::cusp::multiply(::cusp::tbb::par, csr_matrix, x_view, y_view);},
			setting.num_iterations));

		#ifdef ENABLE_VTUNE
		__itt_task_end(domain);
		__itt_pause();
		#endif


		 // GPU cusp
		 cusp::csr_matrix<int, float, cusp::device_memory> d_csr_matrix(csr_matrix);
		 thrust::device_vector<float> d_x(csr_matrix.num_cols, 1);
		 thrust::device_vector<float> d_y(csr_matrix.num_rows, 0);

		 m_result.time.emplace(static_cast<std::string>("cusp_cuda"),
		 	time_spmv(
		 		[&d_csr_matrix, &d_x, &d_y] 
		 		{blas::cusp::spmv(d_csr_matrix, d_x, d_y); throw_on_cuda_error(hipDeviceSynchronize()); },
		 		setting.num_iterations));

		 // GPU cusparse
		 auto cusparse_handle = blas::cusparse::create_handle();
		 auto cusparse_descr = blas::cusparse::create_mat_descr(d_csr_matrix);
		 m_result.time.emplace(static_cast<std::string>("cusparse"),
		 	time_spmv( 
		 		[&cusparse_handle, &cusparse_descr, &d_csr_matrix, &d_x, &d_y] 
		 		{blas::cusparse::spmv(cusparse_handle, cusparse_descr, d_csr_matrix, d_x, d_y); throw_on_cuda_error(hipDeviceSynchronize()); }, 
		 		setting.num_iterations));

		 // GPU cusparse_mp
		 m_result.time.emplace(static_cast<std::string>("cusparse_mp"),
		 	time_spmv([&cusparse_handle, &cusparse_descr, &d_csr_matrix, &d_x, &d_y] 
		 			  {blas::cusparse::spmv_mp(cusparse_handle, cusparse_descr, d_csr_matrix, d_x, d_y); throw_on_cuda_error(hipDeviceSynchronize()); }, 
		 		setting.num_iterations));

		b_result.push_back(m_result);
	}

	return b_result;
}
