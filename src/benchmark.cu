#pragma once
#include "benchmark.h"
#include "wrapper.h"
#include <exception>
#include <yao/cuda/cuda_error.h>
#include <yao/io/binary_io.h>
#include <yao/cusp_ext/convert.h>
#include <yao/time/timer.h>
#include <cusp/csr_matrix.h>

#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>

#include <algorithm>
#include <numeric>


bool is_file_exist(std::string filename)
{
	std::ifstream infile(filename);
	return infile.good();
}

void read_matrix(const matrix_info &matrix, const std::vector<std::string> path,
	cusp::csr_matrix<int, float, cusp::host_memory> &csr_matrix)
{

	std::string filename;
	bool is_found = false;
	for (auto &dir : path)
	{
		filename = dir + matrix.data_filename;
		if (is_file_exist(filename))
		{
			is_found = true;
			break;
		}
	}

	if (!is_found)
		throw(filename + " NOT found!");

	std::vector<float> raw_data;
	yao::io::ReadBinaryArray(filename, raw_data);
	yao::cusp_ext::convert(&raw_data[0], matrix.num_rows, matrix.num_cols, csr_matrix);
	
}

// TODO Reduce redundant code
benchmark_result start_benchmark(benchmark_setting &setting)
{
	benchmark_result b_result;

	for (auto & matrix : setting.matrices)
	{
		
		matrix_result m_result(matrix);

		// CPU SpMV
		cusp::csr_matrix<int, float, cusp::host_memory> csr_matrix;
		read_matrix(matrix, setting.matrix_path, csr_matrix);
		std::vector<float> x(matrix.num_cols, 1);
		std::vector<float> y(matrix.num_rows, 0);
		
		auto t1 = std::chrono::high_resolution_clock::now();
		for (int i = 0; i < setting.num_iterations; i++)
			blas::mkl::spmv(csr_matrix, x, y);
		auto t2 = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double, std::milli> mkl_time = t2 - t1;

		m_result.time.emplace_back(static_cast<std::string>("mkl"), static_cast<double>(mkl_time.count()));
		
		// GPU cusp
		cusp::csr_matrix<int, float, cusp::device_memory> d_csr_matrix(csr_matrix);
		thrust::device_vector<float> d_x(matrix.num_cols, 1);
		thrust::device_vector<float> d_y(matrix.num_rows, 0);

		t1 = std::chrono::high_resolution_clock::now();
		for (int i = 0; i < setting.num_iterations; i++)
		{
			blas::cusp::spmv(d_csr_matrix, d_x, d_y);
			throw_on_cuda_error(hipDeviceSynchronize());
		}
		t2 = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double, std::milli> cusp_time = t2 - t1;

		m_result.time.emplace_back(static_cast<std::string>("cusp"), static_cast<double>(cusp_time.count()));

		// GPU cusparse
		auto handle = blas::cusparse::create_handle();
		auto descr = blas::cusparse::create_mat_descr(d_csr_matrix);

		t1 = std::chrono::high_resolution_clock::now();
		for (int i = 0; i < setting.num_iterations; i++)
		{
			blas::cusparse::spmv(handle, descr, d_csr_matrix, d_x, d_y);
			throw_on_cuda_error(hipDeviceSynchronize());
		}
		t2 = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double, std::milli> cusparse_time = t2 - t1;
		m_result.time.emplace_back(static_cast<std::string>("cusparse"), static_cast<double>(cusparse_time.count()));


		// GPU cusparse cusparse_mp
		t1 = std::chrono::high_resolution_clock::now();
		for (int i = 0; i < setting.num_iterations; i++)
		{
			blas::cusparse::spmv_mp(handle, descr, d_csr_matrix, d_x, d_y);
			throw_on_cuda_error(hipDeviceSynchronize());
		}
		t2 = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double, std::milli> cusparse_mp_time = t2 - t1;
		m_result.time.emplace_back(static_cast<std::string>("cusparse_mp"), static_cast<double>(cusparse_mp_time.count()));

		b_result.push_back(m_result);
	}

	return b_result;
}
